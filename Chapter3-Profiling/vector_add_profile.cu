#include <stdio.h>
#include <hip/hip_runtime.h>

const int DSIZE = 32 * 1048576;

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


__global__ void vector_add(float *d_A, float *d_B, float *d_C, int size){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = idx; i < size;i += gridDim.x * blockDim.x){
            d_C[i] = d_A[i] + d_B[i];
        }
    return ;

}


int main(){
    float *A, *B, *C, *d_A, *d_B, *d_C;
    A = new float[DSIZE]; B = new float[DSIZE]; C = new float[DSIZE];
    for (int i = 0; i < DSIZE; i++){
        A[i] = rand() / (float)RAND_MAX; 
        B[i] = rand() / (float)RAND_MAX; 
        C[i] = rand() / (float)RAND_MAX; 
    }

    hipMalloc(&d_A, DSIZE*sizeof(float)); 
    hipMalloc(&d_B, DSIZE*sizeof(float)); 
    hipMalloc(&d_C, DSIZE*sizeof(float));
    cudaCheckErrors("hipMalloc failure");

    // copy the values from hosty to device
    hipMemcpy(d_A, A, DSIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, DSIZE, hipMemcpyHostToDevice);

    int block  = 1;
    int thread = 1;

    vector_add<<<block, thread>>>(d_A, d_B, d_C, DSIZE);
    cudaCheckErrors("kernel launch failure");

    hipMemcpy(C, d_C, DSIZE, hipMemcpyDeviceToHost);

    cudaCheckErrors("kernel execution failure or hipMemcpy");
    printf("A[0] = %f\n", A[0]);
    printf("B[0] = %f\n", B[0]);
    printf("C[0] = %f\n", C[0]);
    return 0;
}